#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Author(s):
//					A01651517 Pedro Luis González Roa
//					A01703947 Juan Alejandro Alcántara Minaya
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define MAXIMUM 1000000 //1e6
#define THREADS 256
#define BLOCKS	MMIN(32, ((MAXIMUM / THREADS) + 1))

__global__ void is_prime(int*a) {
    int tid = threadIdx.x + (blockIdx.x * blockDim.x) + 2;
    while (tid < MAXIMUM) {
        if (a[tid] == -1) {
            a[tid] = 1;
            for (int i = 2; i < sqrt((double)tid); i++) {
                if (tid % i == 0) {
                    a[tid] = 0;
                    break;
                }
            }
            if (a[tid] == 1) {
                for (int i = 2; tid * i < MAXIMUM; i++) {
                    a[tid * i] = 0;
                }
            }
        }
        tid += blockDim.x * gridDim.x;
    }
}

int main(int argc, char* argv[]) {
    int i, *a, *a_gpu;
    double ms;

    // Memory in cpu
    a = (int *)malloc(sizeof(int) * MAXIMUM);
    for (int j = 0; j < MAXIMUM; j++)
        a[j] = -1;

    // Memory in gpu
    hipMalloc((void**) &a_gpu, sizeof(int)*MAXIMUM);
    hipMemcpy(a_gpu, a, sizeof(int)*MAXIMUM, hipMemcpyHostToDevice);

    printf("Starting...\n");
    ms = 0;

    for (i = 0; i < N; i++) {
        start_timer();

        is_prime<<<BLOCKS, THREADS>>>(a_gpu);

        ms += stop_timer();
    }

    // Copy back
    hipMemcpy(a, a_gpu, sizeof(int)*MAXIMUM, hipMemcpyDeviceToHost);

    printf("Expanding the numbers that are prime to TOP_VALUE:\n");
    for (i = 2; i < 30; i++)
        if (a[i]) printf("%i ", i);
	printf("\n");
	printf("avg time = %.5lf ms\n", (ms / N));

    hipFree(a_gpu);
    free(a);
    return 0;
}
